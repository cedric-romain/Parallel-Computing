#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <string.h>
#include "Runtime_Analysis.h"

#define TARGET_INDEX (row * N + column)
#define N 64        // Dimension: width and height of matrix

__global__ void matrixMultiplyGPU(int *a, int *b, int *result) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int column = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && column < N) {
//        printf("%d,%d;", row, column);
        for (int k = 0; k < N; ++k) {
            result[TARGET_INDEX] = a[row * N + k] * b[k * N + column];
        }
    }
}

void matrixMultiplyCPU(int *a, int *b, int *result) {

    for (int row = 0; row < N; ++row) {
        for (int column = 0; column < N; ++column) {
            for (int k = 0; k < N; ++k) {
                result[TARGET_INDEX] = a[row * N + k] * b[k * N + column];
            }
        }
    }
}

int main() {
    int *a_cpu, *b_cpu, *result_cpu;
    int *a_gpu, *b_gpu, *result_gpu;
    int size = N * N * sizeof(int);

    // Allocate memory on CPU
    a_cpu = (int *) malloc(size);
    b_cpu = (int *) malloc(size);
    result_cpu = (int *) malloc(size);

    // Allocate memory on GPU
    hipMallocManaged(&a_gpu, size);
    hipMallocManaged(&b_gpu, size);
    hipMallocManaged(&result_gpu, size);

    // Initialize Matrices
    for (int row = 0; row < N; ++row) {
        for (int column = 0; column < N; ++column) {
            a_cpu[TARGET_INDEX] = row;
            b_cpu[TARGET_INDEX] = column + 2;
            result_cpu[TARGET_INDEX] = 0;
            a_gpu[TARGET_INDEX] = row;
            b_gpu[TARGET_INDEX] = column + 2;
            result_gpu[TARGET_INDEX] = 0;
        }
    }


    dim3 threads_per_block(16, 16, 1);    // 16 x 16 Block-Threads
    dim3 number_of_blocks((N / threads_per_block.x) + 1,
                          (N / threads_per_block.y) + 1,
                          1);    // Two-Dimensional Grid: z = 1

    auto *timer_gpu = new Runtime_Analysis("TimerGPU");
    timer_gpu->setStart();
    matrixMultiplyGPU <<<number_of_blocks, threads_per_block>>>(a_gpu, b_gpu,
                                                                result_gpu);
    timer_gpu->setEnd();
    std::cout << timer_gpu->print(TimerUnits::microseconds).rdbuf();

    hipDeviceSynchronize();

//    printf("\n");

    auto *timer_cpu = new Runtime_Analysis("TimerCPU");
    timer_cpu->setStart();
    matrixMultiplyCPU(a_cpu, b_cpu, result_cpu);
    timer_cpu->setEnd();
    std::cout << timer_cpu->print(TimerUnits::microseconds).rdbuf();

    bool error = false;


    auto *timer_comparison = new Runtime_Analysis("Timer compare both arrays");
    timer_comparison->setStart();
    for (int row = 0; row < N && !error; ++row) {

        for (int column = 0; column < N && !error; ++column) {

//            printf("Values on \t\tGPU: %d\t\tCPU: %d\t\t\t", result_gpu[TARGET_INDEX], result_cpu[TARGET_INDEX]);
//            printf("at index %d\n", TARGET_INDEX);

            if (result_gpu[TARGET_INDEX] !=
                result_cpu[TARGET_INDEX]) {
                printf("Error in matrix multiplication at position[%d][%d]\n",
                       row, column);
                error = true;
                break;
            }
        }
    }
    timer_comparison->setEnd();
    std::cout << timer_comparison->print(TimerUnits::microseconds).rdbuf();
    if (!error) {
        printf("Success!\n");
    }

    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(result_gpu);

    free(a_cpu);
    free(b_cpu);
    free(result_cpu);

    return 0;
}
