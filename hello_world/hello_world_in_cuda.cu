// sourcecode copy/pasta from https://gpgpu.io/2019/12/07/cuda-hello-world/


#include <hip/hip_runtime.h>
#include <iostream>

// CUDA hello world kernel
__global__ void hello_world() {
    // printf is available for all GPUs with compute capability 2.0 and higher.
    printf("Hello World in CUDA!\n");
}

// program main fuction
int main(int argc, char *argv[]) {
    int exit = 0;

    // GPU hello world
    hello_world<<<1,1>>>();

    // This is not an explicit "flush buffer" function, but it serves that purpose here.
    // Without this call it is very likely there won't be output from the GPU.
    hipDeviceSynchronize();
    // basic CUDA error checking
    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << std::endl;
        exit = 1;
    }
    return exit;
}